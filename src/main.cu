#include "hip/hip_runtime.h"
#include <Simulator.h>
#include <iostream>

#include "utils/math.hpp"
#include <cmath>
#include <functional>

int main()
{
    // std::function<float (float)> f = sinf;

    // auto rebound = [](float x, float s, float e) -> float {
    //     // Your code here to transform 'x' and return the result as a float.
    //     return s + math::mod(x - s, e - s); // Example: doubling the input value.
    // };

    // math::LookUpTable<float> mysin(f, 100, 0.0, 2.0 * M_PI, rebound);

    // std::cout << rebound(-1.0, ) << std::endl;

    // return 0;

    //=== INITIALIZATIONS ===//
    simulator::Simulator Simulator;
    Simulator.init();
    opengl::render::OpenglBuffersManager openglBuffersManager;

    srand(GLOBAL_SEED);

    //=== EXECUTION LOOP ===/
    Simulator.run(&openglBuffersManager);

    //=== EXIT ===/
    return EXIT_SUCCESS;
}
